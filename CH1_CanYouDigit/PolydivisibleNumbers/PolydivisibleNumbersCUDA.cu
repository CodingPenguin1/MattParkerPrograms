// typedef struct {
//     unsigned long long int lo;
//     unsigned long long int hi;
//   } my_uint128;

//   my_uint128 add_uint128 (my_uint128 a, my_uint128 b)
//   {
//     my_uint128 res;
//     res.lo = a.lo + b.lo;
//     res.hi = a.hi + b.hi + (res.lo < a.lo);
//     return res;
//   }

// https://github.com/curtisseizert/CUDA-uint128

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <iostream>

#define SLOTS_PER_BLOCK 1024

#define NUM_THREADS 256
#define NUM_BLOCKS 256

__constant__ uint64_t POWS[19] = {1ll, 10ll, 100ll, 1000ll, 10000ll, 100000ll, 1000000ll,
                                  10000000ll, 100000000ll, 1000000000ll, 10000000000ll,
                                  100000000000ll, 1000000000000ll, 10000000000000ll,
                                  100000000000000ll, 1000000000000000ll, 10000000000000000ll,
                                  100000000000000000ll, 1000000000000000000ll};


__global__ void isPolydivisible(uint64_t maxNum, uint64_t *results) {

    uint64_t start = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    uint16_t numberFound = 0;
    uint64_t *threadResults = results + start * SLOTS_PER_BLOCK;

    for(uint64_t n = start; n < maxNum; n += stride) {
        bool polydivisible = true;
        uint16_t numDigits = 1;

        for(int i = 0; i < 19; ++i) {
            if(n >= POWS[i]) {
                numDigits = i + 1;
            }
        }

        __syncthreads();

        for(int i = 0; i < numDigits; ++i) {
            uint64_t num = n / POWS[i];
            if (num % (numDigits - i) != 0)
                polydivisible = false;
        }

        if(polydivisible)
            threadResults[numberFound++] = n;

        // Maybe sync threads?
    }
}


int main() {

    std::cout << "Enter the maximum number: " << std::flush;
    uint64_t maxNum = 0;
    std::cin >> maxNum;

    uint64_t *results;
    uint64_t arrLen = SLOTS_PER_BLOCK * NUM_BLOCKS * NUM_THREADS;
    hipMallocManaged(&results, arrLen  * sizeof(uint64_t) );

    for(uint64_t i = 0; i < arrLen; ++i) {
        results[i] = 0;
    }

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    isPolydivisible<<<NUM_BLOCKS,NUM_THREADS>>>(maxNum, results);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float millis = 0;
    hipEventElapsedTime(&millis, start, stop);

    uint64_t count = 1;
    for(uint64_t i = 0; i < arrLen; ++i) {
        if(results[i] != 0) {
            std::cout << results[i] << "\n";
            ++count;
        }
    }

    std::cout << std::flush;

    printf("%0.3fs, %d numbers found\n", millis / 1000, count);
}
